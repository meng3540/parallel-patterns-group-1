#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

// Define shared memory section size as a constant
#define SECTION_SIZE 32

hipError_t launch_Kogge_Stone_scan_kernel(float* X, float* Y, unsigned int N);

__global__ void Kogge_Stone_scan_kernel(float* X, float* Y, unsigned int N) {
    __shared__ float XY[SECTION_SIZE];
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < N) {
        XY[threadIdx.x] = X[i];
    }
    else {
        XY[threadIdx.x] = 0.0f;
    }

    for (unsigned int stride = 1; stride < blockDim.x; stride *= 2) {
        __syncthreads();
        float temp;
        if (threadIdx.x >= stride)
            temp = XY[threadIdx.x] + XY[threadIdx.x - stride];
        __syncthreads();
        if (threadIdx.x >= stride)
            XY[threadIdx.x] = temp;
    }

    if (i < N) {
        Y[i] = XY[threadIdx.x];
    }
}

int main() {
    const int arraySize = 5;
    float x[arraySize] = { 1, 2, 3, 4, 5 };
    float y[arraySize] = { 0 };

    // Launch CUDA kernel
    hipError_t cudaStatus = launch_Kogge_Stone_scan_kernel(x, y, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Kogge_Stone_scan_kernel failed!");
        return 1;
    }

    printf("{1,2,3,4,5} => {%f, %f, %f, %f, %f}\n",
        y[0], y[1], y[2], y[3], y[4]);

    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function to allocate memory and launch the kernel
hipError_t launch_Kogge_Stone_scan_kernel(float* x, float* y, unsigned int arraySize) {
    float* dev_x = 0;
    float* dev_y = 0;
    hipError_t cudaStatus;

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_x, arraySize * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_y, arraySize * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_x, x, arraySize * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch CUDA kernel
    Kogge_Stone_scan_kernel << <(arraySize + SECTION_SIZE - 1) / SECTION_SIZE, SECTION_SIZE >> > (dev_x, dev_y, arraySize);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d!\n", cudaStatus);
        goto Error;
    }

    cudaStatus = hipMemcpy(y, dev_y, arraySize * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_x);
    hipFree(dev_y);
    return cudaStatus;
}
