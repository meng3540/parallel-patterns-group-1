#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

/* Define shared memory section size */
#define SECTION_SIZE 32

hipError_t launch_Kogge_Stone_scan_kernel(float* X, float* Y, unsigned int N);

/* CUDA kernel implementing the Kogge-Stone scan algorithm */
__global__ void Kogge_Stone_scan_kernel(float* X, float* Y, unsigned int N) {
    /* Shared memory allocation for storing sums */
    __shared__ float XY[SECTION_SIZE];
    /* Compute global index for each thread */
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    /* Load input data into shared memory */
    if (i < N) {
        XY[threadIdx.x] = X[i];
    }
    else {
        XY[threadIdx.x] = 0.0f; /* Set out-of-bounds threads to zero */
    }

    /* Perform Kogge-Stone parallel prefix sum */
    for (unsigned int stride = 1; stride < blockDim.x; stride *= 2) {
        __syncthreads(); /* Synchronize threads before reading */
        float temp;
        if (threadIdx.x >= stride)
            temp = XY[threadIdx.x] + XY[threadIdx.x - stride];
        __syncthreads(); /* Synchronize threads before writing */
        if (threadIdx.x >= stride)
            XY[threadIdx.x] = temp;
    }

    /* Write results back to global memory */
    if (i < N) {
        Y[i] = XY[threadIdx.x];
    }
}

int main() {
    const int arraySize = 5;
    float x[arraySize] = { 1, 2, 3, 4, 5 }; /* Input array */
    float y[arraySize] = { 0 }; /* Output array */

    hipError_t cudaStatus = launch_Kogge_Stone_scan_kernel(x, y, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Kogge_Stone_scan_kernel failed!");
        return 1;
    }

    /* Print the computed prefix sum */
    printf("{1,2,3,4,5} => {%f, %f, %f, %f, %f}\n",
        y[0], y[1], y[2], y[3], y[4]);

    /* Reset the CUDA device */
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

/* Helper function to allocate memory and launch the kernel */
hipError_t launch_Kogge_Stone_scan_kernel(float* x, float* y, unsigned int arraySize) {
    float* dev_x = 0; /* Device memory for input array */
    float* dev_y = 0; /* Device memory for output array */
    hipError_t cudaStatus;

    /* Set the CUDA device */
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!");
        goto Error;
    }

    /* Allocate device memory for input array */
    cudaStatus = hipMalloc((void**)&dev_x, arraySize * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    /* Allocate device memory for output array */
    cudaStatus = hipMalloc((void**)&dev_y, arraySize * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    /* CUDA events for timing */
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    /* Copy input data from host to device */
    cudaStatus = hipMemcpy(dev_x, x, arraySize * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    /* Record start time */
    hipEventRecord(start);
 
    /* Launch the CUDA kernel */
    hipDeviceSynchronize();
    Kogge_Stone_scan_kernel << <(arraySize + SECTION_SIZE - 1) / SECTION_SIZE, SECTION_SIZE >> > (dev_x, dev_y, arraySize);

    /* Record stop time */
    hipEventRecord(stop);

    /* Check for kernel launch errors */
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d!\n", cudaStatus);
        goto Error;
    }

    /* Copy results from device to host */
    cudaStatus = hipMemcpy(y, dev_y, arraySize * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    /* Print the kernel execution time */
    printf("Kernel execution time: %.3f ms.\n", milliseconds);

    /* Calculate the total amount of data transferred in bytes */
    long totalDataTransferred = (arraySize * sizeof(float) * 2);

    /* Calculate the effective bandwidth in GB/s */
    float effectiveBandwidth = (totalDataTransferred / (milliseconds / 1000.0f)) / 1e9;

    /* Print the effective bandwidth in GB/s */
    printf("Effective bandwidth (GB/s): %.6f GB/s.\n", effectiveBandwidth);

Error:
    /* Free allocated device memory */
    hipFree(dev_x);
    hipFree(dev_y);
    return cudaStatus;
}
