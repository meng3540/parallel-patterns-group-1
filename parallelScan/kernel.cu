#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

// Define shared memory section size as a constant
#define SECTION_SIZE 32

hipError_t launch_Kogge_Stone_scan_kernel(float* X, float* Y, unsigned int N);

__global__ void Kogge_Stone_scan_kernel(float* X, float* Y, unsigned int N) {
    __shared__ float XY[SECTION_SIZE];
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < N) {
        XY[threadIdx.x] = X[i];
    }
    else {
        XY[threadIdx.x] = 0.0f;
    }

    for (unsigned int stride = 1; stride < blockDim.x; stride *= 2) {
        __syncthreads();
        float temp;
        if (threadIdx.x >= stride)
            temp = XY[threadIdx.x] + XY[threadIdx.x - stride];
        __syncthreads();
        if (threadIdx.x >= stride)
            XY[threadIdx.x] = temp;
    }

    if (i < N) {
        Y[i] = XY[threadIdx.x];
    }
}

int main() {
    const int arraySize = 5;
    float x[arraySize] = { 1, 2, 3, 4, 5 };
    float y[arraySize] = { 0 };

    // Launch CUDA kernel
    hipError_t cudaStatus = launch_Kogge_Stone_scan_kernel(x, y, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Kogge_Stone_scan_kernel failed!");
        return 1;
    }

    printf("{1,2,3,4,5} => {%f, %f, %f, %f, %f}\n",
        y[0], y[1], y[2], y[3], y[4]);

    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function to allocate memory and launch the kernel
hipError_t launch_Kogge_Stone_scan_kernel(float* x, float* y, unsigned int arraySize) {
    float* dev_x = 0;
    float* dev_y = 0;
    hipError_t cudaStatus;

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_x, arraySize * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_y, arraySize * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    cudaStatus = hipMemcpy(dev_x, x, arraySize * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Record the start event
    hipEventRecord(start);

 
    // Launch CUDA kernel
    hipDeviceSynchronize();
    Kogge_Stone_scan_kernel << <(arraySize + SECTION_SIZE - 1) / SECTION_SIZE, SECTION_SIZE >> > (dev_x, dev_y, arraySize);
    hipEventRecord(stop);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d!\n", cudaStatus);
        goto Error;
    }

    cudaStatus = hipMemcpy(y, dev_y, arraySize * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    //Print the kernel execution time
    printf("Kernel execution time: %.3f ms.\n", milliseconds);

    // Calculate the total amount of data transferred (in bytes)
    long totalDataTransferred = (arraySize * sizeof(float) * 2);

    // Calculate effective bandwidth in GB/s
    float effectiveBandwidth = (totalDataTransferred / (milliseconds / 1000.0f)) / 1e9;

    // Print effective bandwidth
    printf("Effective bandwidth (GB/s): %.6f GB/s\n", effectiveBandwidth);

Error:
    hipFree(dev_x);
    hipFree(dev_y);
    return cudaStatus;
}
