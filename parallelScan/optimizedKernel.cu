#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

#define SECTION_SIZE 32  // Number of threads per block

hipError_t launch_Kogge_Stone_scan_kernel(float* X, float* Y, unsigned int N);

// Optimized Kogge-Stone Scan Kernel Using Warp Shuffles
__global__ void Kogge_Stone_scan_kernel(float* X, float* Y, unsigned int N) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    int lane = threadIdx.x % warpSize;  // Lane index within a warp

    // Load input from global memory
    float val = (i < N) ? X[i] : 0.0f;

    // Perform warp-level scan using __shfl_up_sync
    for (int stride = 1; stride < warpSize; stride *= 2) {
        float prev = __shfl_up_sync(0xFFFFFFFF, val, stride, warpSize);
        if (lane >= stride) {
            val += prev;
        }
    }

    // Write results back to global memory efficiently
    if (i < N) {
        Y[i] = val;
    }
}

int main() {
    const int arraySize = 5;
    float x[arraySize] = { 1, 2, 3, 4, 5 };
    float y[arraySize];

    hipError_t cudaStatus = launch_Kogge_Stone_scan_kernel(x, y, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Kogge_Stone_scan_kernel failed!\n");
        return 1;
    }

    printf("{1,2,3,4,5} => {%f, %f, %f, %f, %f}\n", y[0], y[1], y[2], y[3], y[4]);
    
    hipDeviceReset();
    return 0;
}

hipError_t launch_Kogge_Stone_scan_kernel(float* x, float* y, unsigned int arraySize) {
    float* dev_x = nullptr;
    float* dev_y = nullptr;
    hipError_t cudaStatus;

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!\n");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_x, arraySize * sizeof(float));
    cudaStatus = hipMalloc((void**)&dev_y, arraySize * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!\n");
        goto Error;
    }

    // **Memory transfer: Copy data from host to device**
    cudaStatus = hipMemcpy(dev_x, x, arraySize * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!\n");
        goto Error;
    }

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    int threadsPerBlock = SECTION_SIZE;
    int blocksPerGrid = (arraySize + threadsPerBlock - 1) / threadsPerBlock;
    
    Kogge_Stone_scan_kernel<<<blocksPerGrid, threadsPerBlock>>>(dev_x, dev_y, arraySize);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    hipDeviceSynchronize();

    // **Coalesced memory transfer: Copy results back from device to host**
    cudaStatus = hipMemcpy(y, dev_y, arraySize * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!\n");
        goto Error;
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Kernel execution time: %.3f ms.\n", milliseconds);

    long totalDataTransferred = (arraySize * sizeof(float) * 2);
    float effectiveBandwidth = (totalDataTransferred / (milliseconds / 1000.0f)) / 1e9;
    printf("Effective bandwidth (GB/s): %.6f GB/s.\n", effectiveBandwidth);

Error:
    hipFree(dev_x);
    hipFree(dev_y);
    return cudaStatus;
}
